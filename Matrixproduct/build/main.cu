#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include "cuda_util.h"
#include <iostream>
#include <cassert>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Matrix.h"

typedef std::chrono::time_point<std::chrono::high_resolution_clock> tpoint;

__global__ void matrixMultiplication(float* matrix1, float* matrix2, float* result, int m1_cols, int m1_rows, int m2_cols, int m2_rows)
{
	if (m1_cols != m2_rows)
	{
		throw std::length_error("False Matrix size! Can't mulitply.");
	}

	int res_X = blockDim.x * blockIdx.x + threadIdx.x;
	int res_Y = blockDim.y * blockIdx.y + threadIdx.y;

	int offset = threadIdx.x % m2_cols;

	for (int k = 0; k < m1_cols; ++k)
	{
		result[threadIdx.x] += matrix1[res_X + k] * matrix2[offset + m2_rows * k];
	}
}


bool initDevice(int& device_handle, int& max_threads_per_block) {

	int deviceCount = 0;
	checkErrorsCuda(hipGetDeviceCount(&deviceCount));

	if (0 == deviceCount) {
		std::cerr << "initDevice() : No CUDA device found." << std::endl;
		return false;
	}

	// one could implement more complex logic here to find the fastest device
	if (deviceCount > 1) {
		std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
	}

	// set the device
	checkErrorsCuda(hipSetDevice(device_handle));

	hipDeviceProp_t device_props;
	checkErrorsCuda(hipGetDeviceProperties(&device_props, device_handle));
	max_threads_per_block = device_props.maxThreadsPerBlock;

	return true;
}


int main (int /*argc*/, char** /*argv*/)
{
	int i = 3, j = 3, k = 3;

	Matrix<float> matrix1_host(i, j);
	Matrix<float> matrix2_host(j, k);
	Matrix<float> result_host(matrix1_host.getRows(), matrix2_host.getCols());

	matrix1_host.fillMatrix();
	matrix1_host.printMatrix();

	matrix2_host.fillMatrix();
	matrix2_host.printMatrix();

	// check execution environment
	int device_handle = 0;
	int max_threads_per_block = 0;
	if (!initDevice(device_handle, max_threads_per_block)) {
		return EXIT_FAILURE;
	}

	// initialize memory
	float* result_device = nullptr;
	float* matrix1_device = nullptr;
	float* matrix2_device = nullptr;

	// allocate device memory
	checkErrorsCuda(hipMalloc((void **)&result_device, sizeof(float) * result_host.getTotalSize()));
	checkErrorsCuda(hipMalloc((void **)&matrix1_device, sizeof(float) * matrix1_host.getTotalSize()));
	checkErrorsCuda(hipMalloc((void **)&matrix2_device, sizeof(float) * matrix2_host.getTotalSize()));

	// copy device memory
	checkErrorsCuda(hipMemcpy((void*)matrix1_device, &matrix1_host.m_ptValues, sizeof(float) * matrix1_host.getTotalSize(),
		hipMemcpyHostToDevice));
	checkErrorsCuda(hipMemcpy((void*)matrix2_device, &matrix2_host.m_ptValues, sizeof(float) * matrix2_host.getTotalSize(),
		hipMemcpyHostToDevice));

	// determine thread layout
	dim3 num_threads_per_block(1, 1, 1);
	dim3 num_blocks(1, 1, 1);

	int max_threads_per_block_sqrt = (int)std::sqrt((double)max_threads_per_block);
	assert(32 == max_threads_per_block_sqrt);

	num_blocks.x = result_host.getCols() / max_threads_per_block_sqrt;
	if (0 != result_host.getCols() % max_threads_per_block_sqrt)
	{
		num_blocks.x++;
	}

	num_blocks.y = result_host.getRows() / max_threads_per_block_sqrt;
	if (0 != result_host.getRows() % max_threads_per_block_sqrt)
	{
		num_blocks.y++;
	}

	num_threads_per_block.x = max_threads_per_block_sqrt;
	num_threads_per_block.y = max_threads_per_block_sqrt;

	// run kernel
	tpoint t_start = std::chrono::high_resolution_clock::now();
	//convSeparable<kernel_supp_half> << < num_blocks, num_threads_per_block >> >(kernel_device, image_device, image_conv_device, image.n_rows);
	matrixMultiplication << <num_blocks, num_threads_per_block >> > (matrix1_device, matrix2_device, result_device, matrix1_host.getCols(), matrix1_host.getRows(), matrix2_host.getCols(), matrix2_host.getRows());

	tpoint t_end = std::chrono::high_resolution_clock::now();
	double wall_clock = std::chrono::duration<double, std::milli>(t_end - t_start).count();
	std::cerr << "Execution time: " << wall_clock << " ms." << std::endl;

	checkLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();

	// copy result back to host
	checkErrorsCuda(hipMemcpy(&result_host.m_ptValues, result_device,	sizeof(float) * result_host.getTotalSize(),
		hipMemcpyDeviceToHost));
}

